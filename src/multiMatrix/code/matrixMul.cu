/***
 * 矩阵乘法CPU和GPU实现
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#define BLOCK_SIZE 16

__global__ void gpu_mul_matrix(int *d_a, int *d_b, int *d_c, int matrix_size)
{
  int y = blockDim.y * blockIdx.y + threadIdx.y; // 矩阵的行位置
  int x = blockDim.x * blockIdx.x + threadIdx.x; // 矩阵的列位置
  if (y < matrix_size && x < matrix_size)
  {
    int tmp = 0;
    for (int step = 0; step < matrix_size; step++)
    {
      tmp = tmp + d_a[y * matrix_size + step] * d_b[step * matrix_size + x];
    }
    d_c[y * matrix_size + x] = tmp;
  }
}

void cpu_mul_matrix(int *h_a, int *h_b, int *h_cc, int matrix_size)
{
  for (int i = 0; i < matrix_size; i++)
  {
    for (int j = 0; j < matrix_size; j++)
    {
      int tmp = 0;
      for (int step = 0; step < matrix_size; step++)
      {
        tmp = tmp + h_a[i * matrix_size + step] * h_b[j + step * matrix_size];
      }
      h_cc[j + i * matrix_size] = tmp;
    }
  }
}

int main()
{
  int matrix_size = 1000;
  int *h_a, *h_b, *h_c, *h_cc;
  hipHostMalloc((void **)&h_a, sizeof(int) * matrix_size * matrix_size, hipHostMallocDefault);
  hipHostMalloc((void **)&h_b, sizeof(int) * matrix_size * matrix_size, hipHostMallocDefault);
  hipHostMalloc((void **)&h_c, sizeof(int) * matrix_size * matrix_size, hipHostMallocDefault);
  hipHostMalloc((void **)&h_cc, sizeof(int) * matrix_size * matrix_size, hipHostMallocDefault);
  for (int y = 0; y < matrix_size; y++)
  {
    for (int x = 0; x < matrix_size; x++)
    {
      h_a[y * matrix_size + x] = rand() % 1024;
      h_b[y * matrix_size + x] = rand() % 1024;
    }
  }
  int *d_a, *d_b, *d_c;
  hipMalloc((void **)&d_a, sizeof(int) * matrix_size * matrix_size);
  hipMalloc((void **)&d_b, sizeof(int) * matrix_size * matrix_size);
  hipMalloc((void **)&d_c, sizeof(int) * matrix_size * matrix_size);

  hipMemcpy(d_a, h_a, sizeof(int) * matrix_size * matrix_size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, sizeof(int) * matrix_size * matrix_size, hipMemcpyHostToDevice);

  unsigned int grid_rows = (matrix_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int grid_cols = (matrix_size + BLOCK_SIZE - 1) / BLOCK_SIZE;

  dim3 dimGrid(grid_rows, grid_cols);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  gpu_mul_matrix<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, matrix_size);
  hipMemcpy(h_c, d_c, sizeof(int) * matrix_size * matrix_size, hipMemcpyDeviceToHost);

  cpu_mul_matrix(h_a, h_b, h_cc, matrix_size);

  bool result_flag = true;
  for (int i = 0; i < matrix_size; i++)
  {
    for (int j = 0; j < matrix_size; j++)
    {
      if (fabs(h_cc[i * matrix_size + j] - h_c[i * matrix_size + j]) > 0.000001)
      {
        std::cout << "error position:" << i * matrix_size + j << std::endl;
        result_flag = false;
      }
    }
  }
  std::cout << "Result: " << result_flag << std::endl;
  hipHostFree(h_a);
  hipHostFree(h_b);
  hipHostFree(h_c);
  hipHostFree(h_cc);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;
}